#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "tempo.h"

#define TAM 30800


//-----aloca memória global - na RAM e na GPU
__managed__ char genoma1[TAM];
__managed__ char genoma2[TAM];
__managed__ int equals[10][10];
 

__global__ void compara(int *quantidade)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ((genoma1[idx] != NULL) || (genoma2[idx] != NULL))
	{
		if (genoma1[idx] == genoma2[idx])
		{
			atomicAdd(quantidade, 1);
		}
	}
	genoma1[idx] = NULL;
	genoma2[idx] = NULL;
}


void equals()
{
    int maior = 0;
    int val1;
    int val2;

    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            if(maior < maior[i][j]){
							maior = maior[i][j];
                val1 = i;
                val2 = j;
						}
						printf(" %5d ", matriz_quantidade_iguais[i][j]);
				}
				printf("\n");
    }
}

int main(int argc, char *argv[0])
{
	tempo1();
	int *num;
	*num = 0;

	int i;

  hipMallocManaged(&num, 4);
  srand(time(NULL));

	for (int j = 0; j < 9; j++)
	{
		for (int k = j + 1; k < 10; k++)
		{
			char g1[14] = "genomas/";
			g1[8] = j + '0';
			strcat(g1, ".txt");

			char g2[14] = "genomas/";
			g2[8] = k + '0';
			strcat(g2, ".txt");
			char c; 

			FILE *file1;
			file1 = fopen(g1, "r");
			int i = 0;
			while ((c = getc(file1)) != EOF)
			{
				gen_a[i] = c;
				i++;
			}
			fclose(file1);

			FILE *file2;
			file2 = fopen(g2, "r");
			i = 0;
			while ((c = getc(file2)) != EOF)
			{
				gen_b[i] = c;
				i++;
			}
			fclose(file2);

			compara<<<700, 44>>>(num);

			hipDeviceSynchronize();

			matriz_quantidade_iguais[j][k] = *num;
			*num = 0;
			printf("\n\n");
		}
	}
	equals();
	tempo2();
	tempoFinal("mili segundos", argv[0], MSGLOG);
}




